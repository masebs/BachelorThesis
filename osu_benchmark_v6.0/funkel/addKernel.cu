// addKernel.cu
// Marc S. Schneider, 2023

#include <hip/hip_runtime.h>

__global__ void addKernel(double* result, const double* var1, const double* var2, const int numElements) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numElements) {
        result[tid] = var1[tid] + var2[tid];
    }
}

#ifdef __cplusplus
extern "C" {
#endif
// Wrapper function for the CUDA kernel
void cudaAdd(const int threadsPerBlock, const int blocksPerGrid, double* d_result, const double* d_var1, const double* d_var2, const int numElements) {
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_result, d_var1, d_var2, numElements);
}
#ifdef __cplusplus
}
#endif
