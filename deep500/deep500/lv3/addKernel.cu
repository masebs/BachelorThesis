// addKernel.cu
// Marc S. Schneider 2023

#include <hip/hip_runtime.h>

#ifdef USE_FLOAT
#define TYPE float
#else
#define TYPE double
#endif

__global__ void addKernel(TYPE* result, const TYPE* var1, const TYPE* var2, const int numElements) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numElements) {
        result[tid] = var1[tid] + var2[tid];
    }
}

#ifdef __cplusplus
extern "C" {
#endif
// Wrapper function for the CUDA kernel
void cudaAdd(const int threadsPerBlock, const int blocksPerGrid, TYPE* d_result, const TYPE* d_var1, const TYPE* d_var2, const int numElements) {
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_result, d_var1, d_var2, numElements);
}
#ifdef __cplusplus
}
#endif
