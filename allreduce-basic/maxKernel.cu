// maxKernel.cu
// Marc S. Schneider 2023

#include <hip/hip_runtime.h>

#ifdef USE_FLOAT
#define TYPE float
#else
#define TYPE double
#endif

__global__ void maxKernel(TYPE* result, const TYPE* var1, const TYPE* var2, const int numElements) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numElements) {
        if (var1[tid] > var2[tid]) 
            result[tid] = var1[tid];
        else 
            result[tid] = var2[tid];
    }
}

#ifdef __cplusplus
extern "C" {
#endif
// Wrapper function for the CUDA kernel
void cudaMax(const int threadsPerBlock, const int blocksPerGrid, TYPE* d_result, const TYPE* d_var1, const TYPE* d_var2, const int numElements) {
    maxKernel<<<blocksPerGrid, threadsPerBlock>>>(d_result, d_var1, d_var2, numElements);
}
#ifdef __cplusplus
}
#endif
